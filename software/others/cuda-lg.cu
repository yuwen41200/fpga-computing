#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <algorithm>
#include <ctime>

using namespace std;

__global__ void blue(unsigned short *mat, int height, int width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x
		, j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < height && j < width) {
		unsigned short mi = mat[i * width + j];
		for (int k = 0; k < 1024; k++)
			mi = mi + (mi >> 3);
		mat[i * width + j] = mi;
	}
}


int Video() {
	// Setup video capture device
	// Link it to the first capture device
	hipError_t err;

	int i, j;
	clock_t cnt = 0, cnt_io = 0;

	unsigned short *hmat[3], *dmat = NULL;
	int rows = 1920, cols = 1080;
	int size = rows * cols * sizeof(unsigned short);
	unsigned short val[3] = { 0, 127, 255 };

	for (int k = 0; k < 3; k++) {
		hmat[k] = (unsigned short *)malloc(size);
		for (i = 0; i < rows; i++)
			for (j = 0; j < cols; j++)
				hmat[k][i * cols + j] = val[k];
	}

	clock_t last = clock();
	for (int k = 0; k < 3; k++) {
		dmat = NULL;
		err = hipMalloc(&dmat, size); if (err != hipSuccess) { puts("Error!"); return 1; }
		err = hipMemcpy(dmat, hmat[k], size, hipMemcpyHostToDevice);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed while Memcpying ! %s\n", hipGetErrorString(err));
			return 1;
		}

		dim3 blk(32, 32);
		dim3 grid((rows + 31) / blk.x, (cols + 31) / blk.y);
		blue << <grid, blk >> >(dmat, rows, cols);
		cnt += clock() - last;
		err = hipGetLastError();
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed launching kernel! %s\n", hipGetErrorString(err));
			return 1;
		}

		err = hipMemcpy(hmat[k], dmat, size, hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed while Memcpying back! %s\n", hipGetErrorString(err));
			return 1;
		}

		hipFree(dmat);

	}

	cnt += clock() - last;


//	cout << endl << "Results from frame 0: " << endl;
//	for (int k = 0; k < 3; ++k)
//		for (int i = 0; i < 256; ++i)
//			for (int j = 0; j < 256; ++j)
//				cout << hmat[k][i * cols + j] << " ";

	printf("Total = %fms\n", 1.0*cnt / (1.0*CLOCKS_PER_SEC / 1000.0));

	for (int k = 0; k < 3; k++) free(hmat[k]);

	return 0;
}

int main()
{
	Video();
	while (1);
	return 0;
}