#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <algorithm>
#include <ctime>

using namespace std;

__global__ void blue(unsigned short *mat, int height, int width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x
		, j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < height && j < width) {
		unsigned short mi = mat[i * width + j];
		for (int k = 0; k < 1024; k++)
			mi = mi + (mi >> 3);
		mat[i * width + j] = mi;
	}
}


int Video() {
	// Setup video capture device
	// Link it to the first capture device
	hipError_t err;

	int i, j;
	clock_t cnt = 0, cnt_io = 0;

	unsigned short *hmat, *dmat = NULL;



	int rows = 256, cols = 256;
	int size = rows * cols * sizeof(unsigned short);
	hmat = (unsigned short *)malloc(size);
	for (i = 0; i < rows; i++)
		for (j = 0; j < cols; j++)
			hmat[i * cols + j] = i * cols + j;


	clock_t last = clock();

	dmat = NULL;
	err = hipMalloc(&dmat, size); if (err != hipSuccess) { puts("Error!"); return 1; }
	err = hipMemcpy(dmat, hmat, size, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed while Memcpying ! %s\n", hipGetErrorString(err));
		return 1;
	}

	dim3 blk(32, 32);
	dim3 grid(rows / blk.x, cols / blk.y);
	blue << <grid, blk >> >(dmat, rows, cols);
	cnt += clock() - last;
	err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed launching kernel! %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMemcpy(hmat, dmat, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed while Memcpying back! %s\n", hipGetErrorString(err));
		return 1;
	}

	cnt += clock() - last;


	cout << endl << "Results from frame 0: " << endl;
	for (int i = 0; i < 256; ++i)
		for (int j = 0; j < 256; ++j)
			cout << hmat[i * cols + j] << " ";

	printf("Total = %fms\n", 1.0*cnt / (1.0*CLOCKS_PER_SEC / 1000.0));

	hipFree(dmat);
	free(hmat);

	return 0;
}

int main()
{
	Video();
	while (1);
	return 0;
}
